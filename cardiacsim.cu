/* 
 * Solves the Panfilov model using an explicit numerical scheme.
 * Based on code orginally provided by Xing Cai, Simula Research Laboratory 
 * and reimplementation by Scott B. Baden, UCSD
 * 
 * Modified and  restructured by Didem Unat, Koc University
 *
 */

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>
#include <stdlib.h>
#include <iostream>
#include <iomanip>
#include <string.h>
#include <math.h>
#include <sys/time.h>
using namespace std;


// Utilities
// 

// Timer
// Make successive calls and take a difference to get the elapsed time.
static const double kMicro = 1.0e-6;
double getTime()
{
    struct timeval TV;
    struct timezone TZ;

    const int RC = gettimeofday(&TV, &TZ);
    if(RC == -1) {
            cerr << "ERROR: Bad call to gettimeofday" << endl;
            return(-1);
    }

    return( ((double)TV.tv_sec) + kMicro * ((double)TV.tv_usec) );

}  // end getTime()

// Allocate a 2D array
double **alloc2D(int m,int n){
   double **E;
   int nx=n, ny=m;
   E = (double**)malloc(sizeof(double*)*ny + sizeof(double)*nx*ny);
   assert(E);
   int j;
   for(j=0;j<ny;j++) 
     E[j] = (double*)(E+ny) + j*nx;
   return(E);
}
    
// Reports statistics about the computation
// These values should not vary (except to within roundoff)
// when we use different numbers of  processes to solve the problem
 double stats(double **E, int m, int n, double *_mx){
     double mx = -1;
     double l2norm = 0;
     int i, j;
     for (j=1; j<=m; j++)
       for (i=1; i<=n; i++) {
	   l2norm += E[j][i]*E[j][i];
	   if (E[j][i] > mx)
	       mx = E[j][i];
      }
     *_mx = mx;
     l2norm /= (double) ((m)*(n));
     l2norm = sqrt(l2norm);
     return l2norm;
 }

// External functions
extern "C" {
    void splot(double **E, double T, int niter, int m, int n);
}
void cmdLine(int argc, char *argv[], double& T, int& n, int& px, int& py, int& plot_freq, int& no_comm, int&num_threads);

__device__
int pos(const int j, const int i, const int m, const int n)
{
    return j*n + i;
}

__global__
void copyGhostRegion(double* Eprev, const int m, const int n)
{
    int leftTo = pos(threadIdx.x+1, 0, m, n);
    int leftFrom = pos(threadIdx.x+1, 2, m, n);
    Eprev[leftTo] = Eprev[leftFrom];

    int rightTo = pos(threadIdx.x+1, n+1, m, n);
    int rightFrom = pos(threadIdx.x+1, n-1, m, n);
    Eprev[rightTo] = Eprev[rightFrom];

    int topTo = pos(0, threadIdx.x+1, m, n);
    int topFrom = pos(2, threadIdx.x+1, m, n);
    Eprev[topTo] = Eprev[topFrom];

    int bottomTo = pos(m+1, threadIdx.x+1, m, n);
    int bottomFrom = pos(m-1, threadIdx.x+1, m, n);
    Eprev[bottomTo] = Eprev[bottomFrom];
}

__global__
void PDE(double* E, double* Eprev, const double alpha, const int m, const int n)
{
    
}

void simulate (double** E,  double** E_prev,double** R,
           double* d_E, double* d_Eprev, double* d_R,
	       const double alpha, const int n, const int m, const double kk,
	       const double dt, const double a, const double epsilon,
	       const double M1,const double  M2, const double b)
{
    int i, j; 
    /* 
     * Copy data from boundary of the computational box 
     * to the padding region, set up for differencing
     * on the boundary of the computational box
     * Using mirror boundaries
     */
    dim3 ghostBlock(128, 1, 1);
    dim3 ghostGrid(m/ghostBlock.x, 1);

    copyGhostRegion<<< ghostGrid, ghostBlock >>>(d_Eprev, m, n);

    hipMemcpy(E_prev+(m+2), d_Eprev, sizeof(double)*(m+2)*(n+2), hipMemcpyDeviceToHost);

    // Solve for the excitation, the PDE
    for (j=1; j<=m; j++){
      for (i=1; i<=n; i++) {
	E[j][i] = E_prev[j][i]+alpha*(E_prev[j][i+1]+E_prev[j][i-1]-4*E_prev[j][i]+E_prev[j+1][i]+E_prev[j-1][i]);
      }
    }
    
    /* 
     * Solve the ODE, advancing excitation and recovery to the
     *     next timtestep
     */
    for (j=1; j<=m; j++){
      for (i=1; i<=n; i++)
	E[j][i] = E[j][i] -dt*(kk* E[j][i]*(E[j][i] - a)*(E[j][i]-1)+ E[j][i] *R[j][i]);
    }
    
    for (j=1; j<=m; j++){
      for (i=1; i<=n; i++)
	R[j][i] = R[j][i] + dt*(epsilon+M1* R[j][i]/( E[j][i]+M2))*(-R[j][i]-kk* E[j][i]*(E[j][i]-b-1));
    }

}

// Main program
int main (int argc, char** argv)
{
  /*
   *  Solution arrays
   *   E is the "Excitation" variable, a voltage
   *   R is the "Recovery" variable
   *   E_prev is the Excitation variable for the previous timestep,
   *      and is used in time integration
   */
  double **E, **R, **E_prev;
  
  // Various constants - these definitions shouldn't change
  const double a=0.1, b=0.1, kk=8.0, M1= 0.07, M2=0.3, epsilon=0.01, d=5e-5;
  
  double T=1000.0;
  int m=200,n=200;
  int plot_freq = 0;
  int px = 1, py = 1;
  int no_comm = 0;
  int num_threads=1; 

  cmdLine( argc, argv, T, n,px, py, plot_freq, no_comm, num_threads);
  m = n;  
  // Allocate contiguous memory for solution arrays
  // The computational box is defined on [1:m+1,1:n+1]
  // We pad the arrays in order to facilitate differencing on the 
  // boundaries of the computation box
  E = alloc2D(m+2,n+2);
  E_prev = alloc2D(m+2,n+2);
  R = alloc2D(m+2,n+2);
  
  int i,j;
  // Initialization
  for (j=1; j<=m; j++)
    for (i=1; i<=n; i++)
      E_prev[j][i] = R[j][i] = 0;
  
  for (j=1; j<=m; j++)
    for (i=n/2+1; i<=n; i++)
      E_prev[j][i] = 1.0;
  
  for (j=m/2+1; j<=m; j++)
    for (i=1; i<=n; i++)
      R[j][i] = 1.0;
  
  double dx = 1.0/n;

  // For time integration, these values shouldn't change 
  double rp= kk*(b+1)*(b+1)/4;
  double dte=(dx*dx)/(d*4+((dx*dx))*(rp+kk));
  double dtr=1/(epsilon+((M1/M2)*rp));
  double dt = (dte<dtr) ? 0.95*dte : 0.95*dtr;
  double alpha = d*dt/(dx*dx);

  cout << "Grid Size       : " << n << endl; 
  cout << "Duration of Sim : " << T << endl; 
  cout << "Time step dt    : " << dt << endl; 
  cout << "Process geometry: " << px << " x " << py << endl;
  if (no_comm)
    cout << "Communication   : DISABLED" << endl;
  
  cout << endl;
  
  // Start the timer
  double t0 = getTime();
  
 
  // Simulated time is different from the integer timestep number
  // Simulated time
  double t = 0.0;
  // Integer timestep number
  int niter=0;
  
  int size = sizeof(double) * (m+2) * (n+2);
  double *d_E, *d_Eprev, *d_R;

  hipMalloc((void**) &d_E, size);
  hipMemcpy(d_E, E+(m+2), size, hipMemcpyHostToDevice);
  hipMalloc((void**) &d_Eprev, size);
  hipMemcpy(d_Eprev, E_prev+(m+2), size, hipMemcpyHostToDevice);
  hipMalloc((void**) &d_R, size);
  hipMemcpy(d_R, R+(m+2), size, hipMemcpyHostToDevice);

  while (t<T) {
    
    t += dt;
    niter++;
 
    simulate(E, E_prev, R, d_E, d_Eprev, d_R, alpha, n, m, kk, dt, a, epsilon, M1, M2, b); 
    
    //swap current E with previous E
    double **tmp = E; E = E_prev; E_prev = tmp;
    
    if (plot_freq){
      int k = (int)(t/plot_freq);
      if ((t - k * plot_freq) < dt){
    //cudaMemcpy(E+(m+2), d_E, size, cudaMemcpyDeviceToHost);
    
	splot(E,t,niter,m+2,n+2);
      }
    }
  }//end of while loop

  double time_elapsed = getTime() - t0;

  double Gflops = (double)(niter * (1E-9 * n * n ) * 28.0) / time_elapsed ;
  double BW = (double)(niter * 1E-9 * (n * n * sizeof(double) * 4.0  ))/time_elapsed;

  cout << "Number of Iterations        : " << niter << endl;
  cout << "Elapsed Time (sec)          : " << time_elapsed << endl;
  cout << "Sustained Gflops Rate       : " << Gflops << endl; 
  cout << "Sustained Bandwidth (GB/sec): " << BW << endl << endl; 

  double mx;
  hipMemcpy(E_prev+(m+2), d_Eprev, size, hipMemcpyDeviceToHost);
  double l2norm = stats(E_prev,m,n,&mx);
  cout << "Max: " << mx <<  " L2norm: "<< l2norm << endl;

  if (plot_freq){
    cout << "\n\nEnter any input to close the program and the plot..." << endl;
    getchar();
  }
  
  hipFree(d_E); hipFree(d_Eprev); hipFree(d_R);
  free (E);
  free (E_prev);
  free (R);
  
  return 0;
}
